#include "hip/hip_runtime.h"
// =============================================================================
// === GPUQREngine/Include/Kernel/Apply/block_apply_chunk.cu ===================
// =============================================================================

// GPUQREngine, Copyright (c) 2013, Timothy A Davis, Sencer Nuri Yeralan,
// and Sanjay Ranka.  All Rights Reserved.
// SPDX-License-Identifier: GPL-2.0+

//------------------------------------------------------------------------------
// block_apply_chunk macro
//------------------------------------------------------------------------------

// A = A - V*T'*V'*A, for a single chunk of N columns of A, starting at column
// j1 and ending at j1+N-1.
//
// This function uses fixed thread geometry and loop unrolling, which requires
// the geometry to be known at compile time for best efficiency.  It is then
// #include'd by the block_apply_x function (block_apply.cu).  The following
// terms are #define'd by each specific version:
//
//      ROW_PANELSIZE    # of row tiles in V and A
//      COL_PANELSIZE    # of column tiles in C and A
//      CBITTYROWS       # of rows in the C bitty block
//      CBITTYCOLS       # of cols in the C bitty block
//      ABITTYROWS       # of rows in the A bitty block
//      ABITTYCOLS       # of cols in the A bitty block
//
// The C bitty must cannot be larger than the A bitty block, since additional
// registers are used to buffer the A matrix while the C bitty block is being
// computed.  These buffer registers are not used while computing with the A
// bitty block, so for some variants of this kernel, they can be overlapped
// with the A bitty block.
//
// The ROW_PANELSIZE, COL_PANELSIZE, ROW_EDGE_CASE, and COL_EDGE_CASE are
// #define'd by the parent file(s) that include this file.  The *_EDGE_CASE
// macros are then #undefined here.  The bitty block dimensions are defined
// below.  This file is #include'd into block_apply.cu.  It is not a standalone
// function.

{

    //--------------------------------------------------------------------------
    // bitty block sizes
    //--------------------------------------------------------------------------

    #if (ROW_PANELSIZE == 3)

        #if (COL_PANELSIZE == 2)

            //------------------------------------------------------------------
            // 3-by-2 block apply
            //------------------------------------------------------------------

            // V is 3-by-1, C is 1-by-2, A is 3-by-2 (in # tiles)
            // 256 threads, each does a 4-by-2 block of C = T'*V'*A
            #define CBITTYROWS      4
            #define CBITTYCOLS      2
            // 384 threads, each does a 4-by-4 block of A = A-V*C
            #define ABITTYROWS      4
            #define ABITTYCOLS      4

        #else

            //------------------------------------------------------------------
            // 3-by-1 block apply
            //------------------------------------------------------------------

            // V is 3-by-1, C is 1-by-1, A is 3-by-1 (in # tiles)
            // 256 threads, each does a 2-by-2 block of C = T'*V'*A
            #define CBITTYROWS      2
            #define CBITTYCOLS      2
            // 384 threads, each does a 2-by-4 block of A = A-V*C
            #define ABITTYROWS      2
            #define ABITTYCOLS      4

        #endif

    #elif (ROW_PANELSIZE == 2)

        #if (COL_PANELSIZE == 2)

            //------------------------------------------------------------------
            // block_apply_2_by_2
            //------------------------------------------------------------------

            // V is 2-by-1, C is 1-by-2, A is 2-by-2 (in # tiles)
            // 256 threads, each does a 4-by-2 block of C = T'*V'*A
            #define CBITTYROWS      4
            #define CBITTYCOLS      2
            // 256 threads, each does a 4-by-4 block of A = A-V*C
            #define ABITTYROWS      4
            #define ABITTYCOLS      4

        #else

            //------------------------------------------------------------------
            // block_apply_2_by_1
            //------------------------------------------------------------------

            // V is 2-by-1, C is 1-by-1, A is 2-by-1 (in # tiles)
            // 256 threads, each does a 2-by-2 block of C = T'*V'*A
            #define CBITTYROWS      2
            #define CBITTYCOLS      2
            // 256 threads, each does a 2-by-4 block of A = A-V*C
            #define ABITTYROWS      2
            #define ABITTYCOLS      4


        #endif

    #else

        #if (COL_PANELSIZE == 2)

            //------------------------------------------------------------------
            // block_apply_1_by_2
            //------------------------------------------------------------------

            // V is 1-by-1, C is 1-by-2, A is 1-by-2 (in # tiles)
            // 256 threads, each does a 4-by-2 block of C = T'*V'*A
            #define CBITTYROWS      2
            #define CBITTYCOLS      4
            // 256 threads, each does a 4-by-2 block of A = A-V*C
            #define ABITTYROWS      2
            #define ABITTYCOLS      4

        #else

            //------------------------------------------------------------------
            // block_apply_1_by_1
            //------------------------------------------------------------------

            // V is 1-by-1, C is 1-by-1, A is 1-by-1 (in # tiles)
            // 256 threads, each does a 2-by-2 block of C = T'*V'*A
            #define CBITTYROWS      2
            #define CBITTYCOLS      2
            // 256 threads, each does a 2-by-2 block of A = A-V*C
            #define ABITTYROWS      2
            #define ABITTYCOLS      2

        #endif

    #endif

    //--------------------------------------------------------------------------
    // matrix sizes and thread geometry
    //--------------------------------------------------------------------------

    // For each outer iteration, C is M-by-N, V is (K+1)-by-M (with an extra
    // row for T), and A is K-by-N.
    #define K           (ROW_PANELSIZE * M)
    #define N           (COL_PANELSIZE * M)

    // threads to use for C=T'*(V'*A)
    #define CTHREADS    ((M * N) / (CBITTYROWS * CBITTYCOLS))

    // threads to use for A=A-V*C
    #define ATHREADS    ((K * N) / (ABITTYROWS * ABITTYCOLS))

    //--------------------------------------------------------------------------
    // bitty blocks for the computation
    //--------------------------------------------------------------------------

    // Each thread owns a bitty block of C for C=T'*V'*A.  The top left entry
    // owned by a thread is C(ic,jc).  Thread 0 does C(0,0), thread 1 does
    // C(1,0) ...
    #define ic          (threadIdx.x % (M/CBITTYROWS))
    #define jc          (threadIdx.x / (M/CBITTYROWS))
    #define MYCBITTYROW(ii) (ii * (M/CBITTYROWS) + ic)
    #define MYCBITTYCOL(jj) (jj * (N/CBITTYCOLS) + jc)

    // Each thread owns a bitty block of A for A=A-V*C, with top left entry
    // A(ia,ja).  Thread 0 does A(0,0), thread 1 does A(0,1), thread 2 does
    // A(0,2), ... so that global memory loads/stores are coallesced across a
    // warp.
    #define ia          (threadIdx.x / (N/ABITTYCOLS))
    #define ja          (threadIdx.x % (N/ABITTYCOLS))
    #define MYABITTYROW(ii) (ii * (K/ABITTYROWS) + ia)
    #define MYABITTYCOL(jj) (jj * (N/ABITTYCOLS) + ja)

    //--------------------------------------------------------------------------
    // loading the A matrix
    //--------------------------------------------------------------------------

    // Each thread loads a set of entries of A defined by iaload and jaload.
    // The first entry loaded by a thread is A(iaload,jaload), and then it
    // loads entries every ACHUNKSIZE rows after that (in the same column
    // jaload).
    #define iaload      (threadIdx.x / N)
    #define jaload      (threadIdx.x % N)
    #define ACHUNKSIZE  (NUMTHREADS / N)
    #define NACHUNKS    CEIL (HALFTILE*N, NUMTHREADS)

    int fjload = j1 + jaload ;

    //--------------------------------------------------------------------------
    // register allocation
    //--------------------------------------------------------------------------

    // C bitty block is no larger than the A bitty block, in both dimensions.
    double rbit [ABITTYROWS][ABITTYCOLS] ;
    double rrow [ABITTYROWS] ;
    double rcol [ABITTYCOLS] ;

    #if (CBITTYCOLS == ABITTYCOLS)
        // the A bitty block is too small to hold the A buffer
        double abuffer [NACHUNKS] ;
        #define rbitA(i) abuffer [i]
    #else
        // use the last column of the A bitty block for the A buffer
        #define rbitA(i) (rbit [i][ABITTYCOLS-1])
    #endif

    //--------------------------------------------------------------------------
    // edge case
    //--------------------------------------------------------------------------

    #ifdef ROW_EDGE_CASE
        // check if a row is inside the front.
        #define INSIDE_ROW(test) (test)
    #else
        // the row is guaranteed to reside inside the frontal matrix.
        #define INSIDE_ROW(test) (1)
    #endif

    #ifdef COL_EDGE_CASE
        // check if a column is inside the front.
        #define INSIDE_COL(test) (test)
    #else
        // the column is guaranteed to reside inside the frontal matrix.
        #define INSIDE_COL(test) (1)
    #endif

    bool aloader = INSIDE_COL (fjload < fn) ;

    //--------------------------------------------------------------------------
    // C = V'*A, where V is now in shared, and A is loaded from global
    //--------------------------------------------------------------------------

    // prefetch the first halftile of A from global to register
    #pragma unroll
    for (int ii = 0 ; ii < NACHUNKS ; ii++)
    {
        rbitA (ii) = 0 ;
    }
    #pragma unroll
    for (int ii = 0 ; ii < NACHUNKS ; ii++)
    {
        int i = ii * ACHUNKSIZE + iaload ;
        if (ii < NACHUNKS-1 || i < HALFTILE)
        {
            int fi = IFRONT (0, i) ;
            if (aloader && INSIDE_ROW (fi < fm))
            {
                rbitA (ii) = glF [fi * fn + fjload] ;
            }
        }
    }

    // The X=V*C computation in the prior iteration reads shC, but the same
    // space is used to load A from the frontal matrix in this iteration.
    __syncthreads ( ) ;

    // clear the C bitty block
    #pragma unroll
    for (int ii = 0 ; ii < CBITTYROWS ; ii++)
    {
        #pragma unroll
        for (int jj = 0 ; jj < CBITTYCOLS ; jj++)
        {
            rbit [ii][jj] = 0 ;
        }
    }

    // C=V'*A for the first tile of V, which is lower triangular
    #define FIRST_TILE
    #include "cevta_tile.cu"
    
    // Subsequent tiles of V are square.  Result is in C bitty block.
    for (int t = 1 ; t < ROW_PANELSIZE ; t++)
    {
        #include "cevta_tile.cu"
    }

    //--------------------------------------------------------------------------
    // write result of C=V'*A into shared, and clear the C bitty block
    //--------------------------------------------------------------------------

    if (CTHREADS == NUMTHREADS || threadIdx.x < CTHREADS)
    {
        #pragma unroll
        for (int ii = 0 ; ii < CBITTYROWS ; ii++)
        {
            int i = MYCBITTYROW (ii) ;
            #pragma unroll
            for (int jj = 0 ; jj < CBITTYCOLS ; jj++)
            {
                int j = MYCBITTYCOL (jj) ;
                shC [i][j] = rbit [ii][jj] ;
                rbit [ii][jj] = 0 ;
            }
        }
    }

    // make sure all of shC is available to all threads
    __syncthreads ( ) ;

    //--------------------------------------------------------------------------
    // C = triu(T)'*C, leaving the result in the C bitty block
    //--------------------------------------------------------------------------

    if (CTHREADS == NUMTHREADS || threadIdx.x < CTHREADS)
    {
        #pragma unroll
        for (int i = 0 ; i < M ; i++)
        {
            #pragma unroll
            for (int ii = 0 ; ii < CBITTYROWS ; ii++)
            {
                int j = MYCBITTYROW (ii) ;
                if (i <= j)
                {
                    rrow [ii] = ST (i,j) ;
                }
            }
            #pragma unroll
            for (int jj = 0 ; jj < CBITTYCOLS ; jj++)
            {
                int j = MYCBITTYCOL (jj) ;
                rcol [jj] = shC [i][j] ;
            }
            #pragma unroll
            for (int ii = 0 ; ii < CBITTYROWS ; ii++)
            {
                int j = MYCBITTYROW (ii) ;
                if (i <= j)
                {
                    #pragma unroll
                    for (int jj = 0 ; jj < CBITTYCOLS ; jj++)
                    {
                        rbit [ii][jj] += rrow [ii] * rcol [jj] ;
                    }
                }                
            }
        }
    }

    // We need syncthreads here because of the write-after-read hazard.  Each
    // thread reads the old C, above, and then C is modified below with the new
    // C, where newC = triu(T)'*oldC.
    __syncthreads ( ) ;

    //--------------------------------------------------------------------------
    // write the result of C = T'*C to shared memory
    //--------------------------------------------------------------------------

    if (CTHREADS == NUMTHREADS || threadIdx.x < CTHREADS)
    {
        #pragma unroll
        for (int ii = 0 ; ii < CBITTYROWS ; ii++)
        {
            int i = MYCBITTYROW (ii) ;
            #pragma unroll
            for (int jj = 0 ; jj < CBITTYCOLS ; jj++)
            {
                int j = MYCBITTYCOL (jj) ;
                shC [i][j] = rbit [ii][jj] ;
            }
        }
    }

    // All threads come here.  We need a syncthreads because
    // shC has been written above and must be read below in A=A-V*C.
    __syncthreads ( ) ;

    //--------------------------------------------------------------------------
    // A = A - V*C
    //--------------------------------------------------------------------------

    if (ATHREADS == NUMTHREADS || threadIdx.x < ATHREADS)
    {

        //----------------------------------------------------------------------
        // clear the A bitty block
        //----------------------------------------------------------------------

        #pragma unroll
        for (int ii = 0 ; ii < ABITTYROWS ; ii++)
        {
            #pragma unroll
            for (int jj = 0 ; jj < ABITTYCOLS ; jj++)
            {
                rbit [ii][jj] = 0 ;
            }
        }

        //----------------------------------------------------------------------
        // X = tril(V)*C, store result into register (rbit)
        //----------------------------------------------------------------------

        #pragma unroll
        for (int p = 0 ; p < M ; p++)
        {
            #pragma unroll
            for (int ii = 0 ; ii < ABITTYROWS ; ii++)
            {
                int i = MYABITTYROW (ii) ;
                if (i >= p)
                {
                    rrow [ii] = shV [1+i][p] ;
                }
            }
            #pragma unroll
            for (int jj = 0 ; jj < ABITTYCOLS ; jj++)
            {
                int j = MYABITTYCOL (jj) ;
                rcol [jj] = shC [p][j] ;
            }
            #pragma unroll
            for (int ii = 0 ; ii < ABITTYROWS ; ii++)
            {
                int i = MYABITTYROW (ii) ;
                if (i >= p)
                {
                    #pragma unroll
                    for (int jj = 0 ; jj < ABITTYCOLS ; jj++)
                    {
                        rbit [ii][jj] += rrow [ii] * rcol [jj] ;
                    }
                }
            }
        }

        //----------------------------------------------------------------------
        // A = A - X, which finalizes the computation A = A - V*(T'*(V'*A))
        //----------------------------------------------------------------------
        
        #if (COL_PANELSIZE == 2)
        
            #pragma unroll
            for (int ii = 0 ; ii < ABITTYROWS ; ii++)
            {
                int i = MYABITTYROW (ii) ;
                int fi = IFRONT (i / M, i % M) ;
                #pragma unroll
                for (int jj = 0 ; jj < ABITTYCOLS ; jj++)
                {
                    int fj = j1 + MYABITTYCOL (jj) ;
                    if (INSIDE_ROW (fi < fm) && INSIDE_COL (fj < fn))
                    {
                        glF [fi * fn + fj] -= rbit [ii][jj] ;
                    }
                }
            }

        #else
        
            #pragma unroll
            for (int ii = 0 ; ii < ABITTYROWS ; ii++)
            {
                int i = MYABITTYROW (ii) ;
                int fi = IFRONT (i / M, i % M) ;
                #pragma unroll
                for (int jj = 0 ; jj < ABITTYCOLS ; jj++)
                {
                    int fj = j1 + MYABITTYCOL (jj) ;
                    if (INSIDE_ROW (fi < fm) && INSIDE_COL (fj < fn))
                    {
                        shV[i][MYABITTYCOL(jj)] = glF[fi*fn+fj] - rbit[ii][jj];
                    }
                    else
                    {
                        shV[i][MYABITTYCOL(jj)] = 0.0;
                    }
                }
            }
    
        #endif
    }

    //--------------------------------------------------------------------------
    // sync
    //--------------------------------------------------------------------------

    // The X=V*C computation in this iteration reads shC, but the same space is
    // used to load A from the frontal matrix in C=V'*A in the next iteration.
    // This final sync also ensures that all threads finish the block_apply
    // at the same time.  Thus, no syncthreads is needed at the start of a
    // subsequent function (the pipelined apply+factorize, for example).

    __syncthreads ( ) ;
}

//------------------------------------------------------------------------------
// undef's
//------------------------------------------------------------------------------

// The following #define's appear above.  Note that FIRST_TILE is not #undef'd
// since that is done by cevta_tile.cu.
#undef CBITTYROWS
#undef CBITTYCOLS
#undef ABITTYROWS
#undef ABITTYCOLS

#undef K
#undef N

#undef CTHREADS
#undef ATHREADS

#undef ic
#undef jc
#undef MYCBITTYROW
#undef MYCBITTYCOL

#undef ia
#undef ja
#undef MYABITTYROW
#undef MYABITTYCOL

#undef iaload
#undef jaload
#undef ACHUNKSIZE
#undef NACHUNKS

#undef rbitA
#undef INSIDE_ROW
#undef INSIDE_COL

// Defined in the parent file that includes this one.  Note that ROW_PANELSIZE
// is not #undef'd, since that is done in the parent.
#undef ROW_EDGE_CASE
#undef COL_EDGE_CASE
